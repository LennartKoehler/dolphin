#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <operations.h>
#include <kernels.h>
#include <thread>
#include <iostream>
#include <omp.h>

#ifdef ENABLE_CUBE_DEBUG
#define DEBUG_LOG(msg) std::cout << msg << std::endl
#else
#define DEBUG_LOG(msg) // Nichts tun
#endif

namespace CUBE_MAT {
    // Normal Matrix Multiplication
    hipError_t complexMatMulFftwComplexCPU(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        double start = omp_get_wtime();

        // Iterate over the 3D matrix
        for (int x = 0; x < Nx; ++x) {
            for (int y = 0; y < Ny; ++y) {
                for (int z = 0; z < Nz; ++z) {
                    int index = x * Ny * Nz + y * Nz + z; // Correct index for 3D matrix

                    fftw_complex sum = {0.0, 0.0};

                    // Perform matrix multiplication with summation over k
                    for (int k = 0; k < Nz; ++k) {  // Loop over k for multiplication
                        int indexA = x * Ny * Nz + y * Nz + k;
                        int indexB = k * Ny * Nz + y * Nz + z;

                        float realA = A[indexA][0];
                        float imagA = A[indexA][1];
                        float realB = B[indexB][0];
                        float imagB = B[indexB][1];

                        // Accumulate the real and imaginary parts
                        sum[0] += realA * realB - imagA * imagB;
                        sum[1] += realA * imagB + imagA * realB;
                    }

                    // Store the result
                    C[index][0] = sum[0];
                    C[index][1] = sum[1];
                }
            }
        }

        double end = omp_get_wtime();
        DEBUG_LOG("[TIME][" << (end - start) * 1000 << " ms] MatMul in Cpp");
        
        return hipSuccess;
    }
    hipError_t complexMatMulFftwComplexOmpCPU(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        double start = omp_get_wtime();

        // Use OpenMP to parallelize the 3D matrix multiplication
#pragma omp parallel for collapse(3) schedule(static)
        for (int x = 0; x < Nx; ++x) {
            for (int y = 0; y < Ny; ++y) {
                for (int z = 0; z < Nz; ++z) {
                    int index = x * Ny * Nz + y * Nz + z; // Correct index for 3D matrix

                    fftw_complex sum = {0.0, 0.0};

                    // Perform matrix multiplication with summation over k
                    for (int k = 0; k < Nz; ++k) {  // Loop over k for multiplication
                        int indexA = x * Ny * Nz + y * Nz + k;
                        int indexB = k * Ny * Nz + y * Nz + z;

                        float realA = A[indexA][0];
                        float imagA = A[indexA][1];
                        float realB = B[indexB][0];
                        float imagB = B[indexB][1];

                        // Accumulate the real and imaginary parts
                        sum[0] += realA * realB - imagA * imagB;
                        sum[1] += realA * imagB + imagA * realB;
                    }

                    // Store the result
                    C[index][0] = sum[0];
                    C[index][1] = sum[1];
                }
            }
        }

        double end = omp_get_wtime();
        DEBUG_LOG("[TIME][" << (end - start) * 1000 << " ms] MatMul in Cpp with Omp ("
                  << omp_get_max_threads() << " Threads)");
        
        return hipSuccess;
    }
    hipError_t complexMatMulFftwComplexCUDA(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);


        hipEventRecord(start);

        complexMatMulFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);

        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] MatMul in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexMatMulCuComplexCUDA(int Nx, int Ny, int Nz, hipComplex* A, hipComplex* B, hipComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D for Nx * Ny * Nz matrix stored in a 1D array
        dim3 threadsPerBlock(10, 10, 10); // Optimal for testing, adjust for your hardware
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        // Start the event for timing
        hipEventRecord(start);

        // Launch the kernel
        complexMatMulCuComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);

        // Synchronize the device and check for errors after kernel execution
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Check for any CUDA errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        // Calculate and print the time elapsed
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] MatMul in CUDA (hipComplex) ("
                  << threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z << "x"
                  << blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z << ")");
        
        return hipSuccess;
    }
    hipError_t complexMatMulFftwComplex(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C, const char* type) {
        if (!A || !B || !C || !type) {
            return hipErrorInvalidValue;
        }
        
        if (strcmp(type, "cpp") == 0) {
            return complexMatMulFftwComplexCPU(Nx, Ny, Nz, A, B, C);
        }else if (strcmp(type, "omp") == 0) {
            return complexMatMulFftwComplexOmpCPU(Nx, Ny, Nz, A, B, C);
        }else if (strcmp(type, "cuda") == 0) {
            return complexMatMulFftwComplexCUDA(Nx, Ny, Nz, A, B, C);
        }
        else {
            return hipErrorInvalidValue;
        }
    }

    // Elementwise Matrix Multiplication/Division (always GPU)
    hipError_t complexElementwiseMatMulCuComplex(int Nx, int Ny, int Nz, hipComplex* A, hipComplex* B, hipComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatMulCuComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatMul in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatMulCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* A, hipfftComplex* B, hipfftComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);



        complexElementwiseMatMulCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatMul in CUDA (hipfftComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatMulFftwComplex(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);
        hipError_t err = hipGetLastError();

        hipEventRecord(start);

        complexElementwiseMatMulFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);

        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);


        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatMul in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t complexElementwiseMatMulConjugateCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* A, hipfftComplex* B, hipfftComplex* C)  {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatMulConjugateCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatMul conjugated in CUDA (hipfftComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatMulConjugateFftwComplex(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C)  {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatMulConjugateFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatMul conjugated in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatDivCuComplex(int Nx, int Ny, int Nz, hipComplex* A, hipComplex* B, hipComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivCuComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatDiv in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatDivCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* A, hipfftComplex* B, hipfftComplex* C, double epsilon) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatDiv in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatDivFftwComplex(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C, double epsilon) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise MatDiv in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }


    hipError_t complexElementwiseMatDivNaiveCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* A, hipfftComplex* B, hipfftComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivNaiveCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise naive MatDiv in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatDivStabilizedCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* A, hipfftComplex* B, hipfftComplex* C, double epsilon){
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivStabilizedCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise stabilized MatDiv in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
    hipError_t complexElementwiseMatDivStabilizedFftwComplex(int Nx, int Ny, int Nz, fftw_complex* A, fftw_complex* B, fftw_complex* C, double epsilon){
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        complexElementwiseMatDivStabilizedFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return err;
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] elementwise stabilized MatDiv in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return hipSuccess;
    }
}

namespace CUBE_REG {
    // Regularization
    hipError_t calculateLaplacianCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* Afft, hipfftComplex* laplacianfft) {
        if (!Afft || !laplacianfft) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        calculateLaplacianCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, Afft, laplacianfft);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating Laplacian in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradXCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* image, hipfftComplex* gradX) {
        if (!image || !gradX) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientXCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradX);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientX in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradYCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* image, hipfftComplex* gradY) {
        if (!image || !gradY) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientYCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradY);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientY in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradZCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* image, hipfftComplex* gradZ) {
        if (!image || !gradZ) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientZCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradZ);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientZ in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t computeTVCufftComplex(int Nx, int Ny, int Nz, double lambda, hipfftComplex* gx, hipfftComplex* gy, hipfftComplex* gz, hipfftComplex* tv) {
        if (!gx || !gy || !gz || !tv) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        computeTVCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, lambda, gx, gy, gz, tv);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating Total Variation in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t normalizeTVCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* gradX, hipfftComplex* gradY, hipfftComplex* gradZ, double epsilon) {
        if (!gradX || !gradY || !gradZ) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        normalizeTVCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, gradX, gradY, gradZ, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] normalizing Total Variation in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t calculateLaplacianFftwComplex(int Nx, int Ny, int Nz, fftw_complex* Afft, fftw_complex* laplacianfft) {
        if (!Afft || !laplacianfft) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        calculateLaplacianFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, Afft, laplacianfft);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating Laplacian in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradXFftwComplex(int Nx, int Ny, int Nz, fftw_complex* image, fftw_complex* gradX) {
        if (!image || !gradX) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientXFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradX);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientX in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradYFftwComplex(int Nx, int Ny, int Nz, fftw_complex* image, fftw_complex* gradY) {
        if (!image || !gradY) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientYFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradY);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientY in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t gradZFftwComplex(int Nx, int Ny, int Nz, fftw_complex* image, fftw_complex* gradZ) {
        if (!image || !gradZ) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        gradientZFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, image, gradZ);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating GradientZ in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t computeTVFftwComplex(int Nx, int Ny, int Nz, double lambda, fftw_complex *gx, fftw_complex *gy, fftw_complex *gz, fftw_complex *tv) {
        if (!gx || !gy || !gz || !tv) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        computeTVFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, lambda, gx, gy, gz, tv);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating Total Variation in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
    hipError_t normalizeTVFftwComplex(int Nx, int Ny, int Nz, fftw_complex* gradX, fftw_complex* gradY, fftw_complex* gradZ, double epsilon) {
        if (!gradX || !gradY || !gradZ) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        normalizeTVFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, gradX, gradY, gradZ, epsilon);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] normalizing Total Variation in CUDA (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
}

namespace CUBE_TILED {
    // Tiled Memory in GPU
    hipError_t calculateLaplacianCufftComplexTiled(int Nx, int Ny, int Nz, hipfftComplex* Afft, hipfftComplex* laplacianfft) {
        if (!Afft || !laplacianfft) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        calculateLaplacianCufftComplexTiledGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, Afft, laplacianfft);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] calculating Laplacian in CUDA tiled with shared mem (hipComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
}

namespace CUBE_FTT {
    // FFT
    hipError_t cufftForward(hipfftComplex* input, hipfftComplex* output, hipfftHandle plan) {
        if (!input || !output || !plan) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipfftResult result;

        hipEventRecord(start);

        result = hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Forward FFT in cuFFT");
        
        return err; 
    }
    hipError_t cufftInverse(int Nx, int Ny, int Nz, hipfftComplex* input, hipfftComplex* output, hipfftHandle plan) {
        if (!input || !output || !plan) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipfftResult result;

        hipEventRecord(start);

        result = hipfftExecC2C(plan, input, output, HIPFFT_BACKWARD);
        int num_elements = Nx * Ny * Nz;  // Beispiel: Gesamtzahl der Elemente
        int block_size = 256;  // Blockgröße (kann angepasst werden)
        int num_blocks = (num_elements + block_size - 1) / block_size;  // Berechne die Anzahl der Blöcke

        normalizeComplexData<<<num_blocks, block_size>>>(Nx, Ny, Nz, output);

        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Inverse FFT in cuFFT");
        
        return err;
    }

    // Fourier Shift, Padding and Normalization
    hipError_t octantFourierShiftFftwComplexCPU(int Nx, int Ny, int Nz, fftw_complex* data) {
        if (!data) {
            return hipErrorInvalidValue;
        }

        int width = Nx;
        int height = Ny;
        int depth = Nz;
        auto start = std::chrono::high_resolution_clock::now();

        int halfWidth = width / 2;
        int halfHeight = height / 2;
        int halfDepth = depth / 2;

        // Parallelize the nested loops using OpenMP with collapsing to reduce overhead
#pragma omp parallel for collapse(3)
        for (int z = 0; z < halfDepth; ++z) {
            for (int y = 0; y < height; ++y) {
                for (int x = 0; x < width; ++x) {
                    // Calculate the indices for the swap
                    int idx1 = z * height * width + y * width + x;
                    int idx2 = ((z + halfDepth) % depth) * height * width + ((y + halfHeight) % height) * width + ((x + halfWidth) % width);

                    // Perform the swap only if the indices are different
                    if (idx1 != idx2) {
                        // Swap real parts
                        double temp_real = data[idx1][0];
                        data[idx1][0] = data[idx2][0];
                        data[idx2][0] = temp_real;

                        // Swap imaginary parts
                        double temp_imag = data[idx1][1];
                        data[idx1][1] = data[idx2][1];
                        data[idx2][1] = temp_imag;
                    }
                }
            }
        }
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        float time = duration.count();

        DEBUG_LOG("[TIME]["<<time/1000000<<" ms] Octant(Fourier)Shift in CPP");
        
        return hipSuccess;
    }
    hipError_t octantFourierShiftFftwComplex(int Nx, int Ny, int Nz, fftw_complex* data) {
        if (!data) {
            return hipErrorInvalidValue;
        }

        //size_t freeMem, totalMem;
        //hipMemGetInfo(&freeMem, &totalMem);
        //std::cout << "Free memory: " << freeMem << " Total memory: " << totalMem << std::endl;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(2, 2, 2); //=6 //TODO with more threads artefacts visible
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);


        hipEventRecord(start);

        octantFourierShiftFftwComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, data);
        hipError_t errp = hipPeekAtLastError();
        
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Octant(Fouriere)Shift in CUDA (fftw_complex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return (errp == hipSuccess) ? err : errp;
    }
    hipError_t octantFourierShiftCufftComplex(int Nx, int Ny, int Nz, hipfftComplex* data) {
        if (!data) {
            return hipErrorInvalidValue;
        }

        //size_t freeMem, totalMem;
        //hipMemGetInfo(&freeMem, &totalMem);
        //std::cout << "Free memory: " << freeMem << " Total memory: " << totalMem << std::endl;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(2, 2, 2); //=6 //TODO with more threads artefacts visible
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);


        hipEventRecord(start);

        octantFourierShiftCufftComplexGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, data);
        hipError_t errp = hipPeekAtLastError();
        
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Octant(Fouriere)Shift in CUDA (hipfftComplex) ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return (errp == hipSuccess) ? err : errp;
    }
    hipError_t padFftwMat(int oldNx, int oldNy, int oldNz, int newNx, int newNy, int newNz, fftw_complex* oldMat, fftw_complex* newMat)
    {
        if (!oldMat || !newMat) {
            return hipErrorInvalidValue;
        }
        
        auto start = std::chrono::high_resolution_clock::now();
        // Sicherheitsprüfung: Neue Dimensionen müssen größer oder gleich den alten sein
        if (newNx < oldNx || newNy < oldNy || newNz < oldNz) {
            return hipErrorInvalidValue;
        }

        // Offset für Padding (Startkoordinaten der alten Matrix in der neuen Matrix)
        int offsetX = (newNx - oldNx) / 2;
        int offsetY = (newNy - oldNy) / 2;
        int offsetZ = (newNz - oldNz) / 2;

        // Initialisiere die neue Matrix mit Nullen
#pragma omp parallel for
        for (int i = 0; i < newNx * newNy * newNz; ++i) {
            newMat[i][0] = 0.0; // Realteil
            newMat[i][1] = 0.0; // Imaginärteil
        }

        // Kopiere die Werte der alten Matrix in die Mitte der neuen Matrix
#pragma omp parallel for
        for (int z = 0; z < oldNz; ++z) {
            for (int y = 0; y < oldNy; ++y) {
                for (int x = 0; x < oldNx; ++x) {
                    // Index in der alten Matrix
                    int oldIndex = z * oldNy * oldNx + y * oldNx + x;

                    // Index in der neuen Matrix
                    int newIndex =
                        (z + offsetZ) * newNy * newNx +
                        (y + offsetY) * newNx +
                        (x + offsetX);

                    // Kopiere den Wert
                    newMat[newIndex][0] = oldMat[oldIndex][0]; // Realteil
                    newMat[newIndex][1] = oldMat[oldIndex][1]; // Imaginärteil
                }
            }
        }
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        float time = duration.count();

        DEBUG_LOG("[TIME]["<<time/1000000<<" ms] padded FftwComplex Mat in CPP");
        
        return hipSuccess;
    }
    hipError_t padCufftMat(int oldNx, int oldNy, int oldNz, int newNx, int newNy, int newNz, hipfftComplex* d_oldMat, hipfftComplex* d_newMat)
    {
        // Sicherheitsprüfung: Neue Dimensionen müssen größer oder gleich den alten sein
        if (newNx < oldNx || newNy < oldNy || newNz < oldNz) {
            return hipErrorInvalidValue;
        }

        // Offset berechnen
        int offsetX = (newNx - oldNx) / 2;
        int offsetY = (newNy - oldNy) / 2;
        int offsetZ = (newNz - oldNz) / 2;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Block- und Grid-Dimensionen festlegen
        dim3 blockDim(16, 8, 8); // Größe der Blöcke
        dim3 gridDim(
            (newNx + blockDim.x - 1) / blockDim.x,
            (newNy + blockDim.y - 1) / blockDim.y,
            (newNz + blockDim.z - 1) / blockDim.z);

        hipEventRecord(start);
        padCufftMatGlobal<<<gridDim, blockDim>>>(
            oldNx, oldNy, oldNz,
            newNx, newNy, newNz,
            d_oldMat, d_newMat,
            offsetX, offsetY, offsetZ);

        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] padded CufftComplex Mat in CUDA ("<<blockDim.x*blockDim.y*blockDim.z<<"x"<<gridDim.x*gridDim.y*gridDim.z<<")");
        
        return err;
    }
    hipError_t normalizeFftwComplexData(int Nx, int Ny, int Nz, fftw_complex* d_data) {
        if (!d_data) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int num_elements = Nx * Ny * Nz;  // Beispiel: Gesamtzahl der Elemente
        int block_size = 1024;
        int num_blocks = (num_elements + block_size - 1) / block_size;

        hipEventRecord(start);

        normalizeFftwComplexDataGlobal<<<num_blocks, block_size>>>(Nx, Ny, Nz, d_data);
        hipError_t errp = hipPeekAtLastError();
        
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Normalizing FFT data in CUDA (fftw_complex) ("<<block_size*num_blocks<< " Threads)");
        
        return (errp == hipSuccess) ? err : errp;
    }
}

namespace CUBE_DEVICE_KERNEL {
    // Testing __device__ kernels
    hipError_t deviceTestKernel(int Nx, int Ny, int Nz, hipComplex* A, hipComplex* B, hipComplex* C) {
        if (!A || !B || !C) {
            return hipErrorInvalidValue;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Kernel dimension 3D, because 3D matrix stored in 1D array, index in kernel operation depend on structure
        dim3 threadsPerBlock(10, 10, 10); //=1000 (faster than max 1024)
        dim3 blocksPerGrid((Nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (Ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                           (Nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

        hipEventRecord(start);

        deviceTestKernelGlobal<<<blocksPerGrid, threadsPerBlock>>>(Nx, Ny, Nz, A, B, C);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipGetLastError();
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        DEBUG_LOG("[TIME][" << milliseconds << " ms] Device kernel(s) finished ("<<threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z<<"x"<<blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z<<")");
        
        return err;
    }
}






