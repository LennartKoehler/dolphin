#include "kernels.h"
#include <hip/hip_complex.h>

/*
 //TODO
__device__ void complexMatMulCuComplexDevice(int N, hipComplex* A, hipComplex* B, hipComplex* C) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < N && y < N && z < N) {
        int index = x * N * N + y * N + z;
        hipComplex sum = make_hipComplex(0.0f, 0.0f);

        for (int k = 0; k < N; ++k) {
            int indexA = x * N * N + y * N + k;
            int indexB = k * N * N + y * N + z;
            sum = hipCaddf(sum, hipCmulf(A[indexA], B[indexB]));
        }

        C[index] = sum;
    }
}*/

